// https://wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake
// Justin Francis


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;
    float min_cc = 5.0; // TODO: figure out what this should be.

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc; /* Failure */
    }
    if((dP.major+(dP.minor/10)) < min_cc) {
        printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not Building CUDA Code",
               min_cc, dP.major, dP.minor);
        return 1; /* Failure */
    } else {
        printf("sm_%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}