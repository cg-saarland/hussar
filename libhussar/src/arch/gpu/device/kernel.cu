#include "hip/hip_runtime.h"
#pragma diag_suppress 3126
#define RT_FUNCTION static __forceinline__ __device__

#include "kernel.h"
#include <hussar/hussar.h>

#include <optix_device.h>
#include <utility>

using namespace hussar;

#include "utils.h"
#include "polyfills.h"
#include "vecmath.h"

#define log(...)
//#define log(...) printf(__VA_ARGS__)

extern "C" {
  __constant__ GPUParams params;
}

struct RT {
    bool visible(Intersection &isect) const {
        unsigned int visible = 1u;

        optixTrace(
            params.handle,
            vec3_to_float3(isect.ray.o),
            vec3_to_float3(isect.ray.d),
            Epsilon,
            isect.tMax,
            0.0f,                    // rayTime
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            RAY_TYPE_OCCLUSION,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            RAY_TYPE_OCCLUSION,      // missSBTIndex
            visible
        );
        
        return visible;
    }

    void intersect(Intersection &isect) const {
        unsigned int u0, u1;
        packPointer(&isect, u0, u1);

        optixTrace(
            params.handle,
            vec3_to_float3(isect.ray.o),
            vec3_to_float3(isect.ray.d),
            Epsilon,
            isect.tMax,
            0.0f,                     // rayTime
            OptixVisibilityMask(1),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,        // SBT offset
            RAY_TYPE_COUNT,           // SBT stride
            RAY_TYPE_RADIANCE,        // missSBTIndex
            u0, u1
        );
    }
};

extern "C" __global__ void __raygen__rg() {
    const uint3 optixIndex = optixGetLaunchIndex();
    const long sampleIndex = optixIndex.y * params.width + optixIndex.x + params.offset;

    const Scene &scene = *params.d_scene;
    PathTracer &integrator = *params.d_integrator;

    integrator.sample(scene, RT {}, sampleIndex);
}

extern "C" __global__ void __closesthit__radiance() {
  log("  closesthit_radiance\n");

  HitGroupData *rt_data = (HitGroupData *)optixGetSbtDataPointer();

  const TriangleMesh::IndexTriplet &indices = rt_data->indices[optixGetPrimitiveIndex()];
  const float3 v0 = vec3_to_float3(rt_data->vertices[indices.v0]);
  const float3 v1 = vec3_to_float3(rt_data->vertices[indices.v1]);
  const float3 v2 = vec3_to_float3(rt_data->vertices[indices.v2]);

  float3 normal = normalize(cross(v1 - v0, v2 - v0));

  // transform normal to world coordinates
  // float4 worldToObject[3];
  // optix_impl::optixGetWorldToObjectTransformMatrix(worldToObject[0], worldToObject[1], worldToObject[2]);
  // normal = normalize(optix_impl::optixTransformNormal(worldToObject[0], worldToObject[1], worldToObject[2], normal));
  
  // write intersection data
  Intersection &isect = getIsect();
  isect.t = optixGetRayTmax();
  isect.p = float3_to_vec3(optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection());
  isect.n = float3_to_vec3(faceforward(normal, -optixGetWorldRayDirection(), normal));
}

extern "C" __global__ void __miss__radiance() {
    log("  miss_radiance\n");
    // nothing to do
}

extern "C" __global__ void __closesthit__occlusion() {
    optixSetPayload_0(0u); // set visible payload to 0
}
